/*
 *  model.c
 *
 *
 */


#include <math.h>
#include <hip/hip_runtime.h>

struct model_data_
{
	double mygamma;
        double *theta;
        int N_samples;
        int N_sensors;
} model_data;


extern "C"
__global__ void GPU_model(double *g, double *d,double *theta,double mygamma,int N_samples,int N_sensors)
{

	 int ix= blockDim.x*blockIdx.x+threadIdx.x;
	 int iy= blockDim.y*blockIdx.y+threadIdx.y;

	 if(ix<N_samples && iy<N_sensors)
	 {
      g[ix*N_sensors+iy] = 0.0;

      g[ix*N_sensors+iy] = mygamma*theta[ix*2+1]/(2*M_PI*(pow((d[0]+iy*d[1])-theta[ix*2+0],2.0) + pow(theta[ix*2+1],2)));

	 }
}


extern "C"
void model(double *g, double *d) 
{
	double mygamma = model_data.mygamma;
	double *theta = model_data.theta;
	int N_samples = model_data.N_samples;
	int N_sensors = model_data.N_sensors;
	//////////////////////////////////
	// model.m
	// This function provides the model function (= velocity measurements) for the 
	// vortex optimal sensor placement problem.
	// Input:
	// mygamma     : vortex strength
	// theta     : samples of model parameters
	// d         : vector of design parameters, d = [x_s, h]
	// N_sensors : number of sensors to be placed
	//
	// Author: Franziska Krummenacher, krummenf@student.ethz.ch
	// Spring/Summer 2016
	//////////////////////////////////

	//extract number of samples
	//N_samples = size(theta,1);

	//initialize output matrix
	//g = zeros(N_samples,N_sensors);

////allocate GPU  memory 

	double *d_g;
	double *d_d;
	double *d_theta;

	hipMalloc(&d_g,N_samples*N_sensors*sizeof(double));
	hipMalloc(&d_d,2*sizeof(double));
	hipMalloc(&d_theta,2*N_samples*sizeof(double));
	//GPU memory copy
	hipMemcpy(d_g,g,N_samples*N_sensors*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_d,d,2*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_theta,theta,2*N_samples*sizeof(double),hipMemcpyHostToDevice);
	//kernel call 
    
    dim3 blocks(40,1);
    dim3 threads(100,5);
	GPU_model<<<blocks,threads>>>(d_g,d_d,d_theta,mygamma,N_samples,N_sensors);

    hipMemcpy(g,d_g,N_samples*N_sensors*sizeof(double),hipMemcpyDeviceToHost);

    hipFree(d_g);
    hipFree(d_d);
    hipFree(d_theta);

	//for (int i = 0; i < N_samples; i++)
	//	for (int j = 0; j < N_sensors; j++)
	//		g[i*N_sensors+j] = 0.0;

	//fill in model predictions
	//for (int i = 0; i < N_samples; i++)
	//	for (int k = 0; k < N_sensors; k++)
	//	{
	//		g[i*N_sensors+k] = mygamma*theta[i*2+1]/(2*M_PI*(pow((d[0]+k*d[1])-theta[i*2+0],2.0) + pow(theta[i*2+1],2)));
	//	}

	return;
}
